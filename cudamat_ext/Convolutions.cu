#include <hip/hip_runtime.h>  
#include "Convolutions_kernels.cuh"
#include "cudamat.cuh" 

extern "C" 
{

inline bool checkCUDAError() 
{
    hipError_t err = hipGetLastError();

    //if (hipSuccess != err)
        //printf("%s\n", hipGetErrorString( err));
    return hipSuccess != err;
}

extern int ShiftedConvolution(cudamat *signal1, cudamat *signal2, cudamat *target, int kernelWidth, 
    cudamat *scratchPad)
{

   if (!signal1->on_device || !target->on_device || !signal2->on_device || !scratchPad->on_device)
      return ERROR_NOT_ON_DEVICE;

   if (signal1->size[0] != 1 && signal1->size[1] != 1)
      return ERROR_INCOMPATIBLE_DIMENSIONS ;

   int signalLength = signal1->size[0] * signal1->size[1] ; 
   int numKernels = signal2->size[1] ; 

   if (signal2->size[0] != signalLength)
      return ERROR_INCOMPATIBLE_DIMENSIONS ;

   if (target->size[0] != kernelWidth || target->size[1] != numKernels)
      return ERROR_INCOMPATIBLE_DIMENSIONS;

   if (signal1->is_trans)
      return ERROR_TRANSPOSED;
   if (signal2->is_trans)
      return ERROR_TRANSPOSED;
   if (target->is_trans)
      return ERROR_TRANSPOSED;

   // Do calculation on device:  
   int numThreadsPerBlock = 256 ;  
   const int numPtsPerBlock = 512 ;

   int numBlocksPerKernel = signalLength/numPtsPerBlock + (signalLength%numPtsPerBlock == 0 ? 0:1); 
   if (scratchPad->size[0]*scratchPad->size[1] < kernelWidth*numBlocksPerKernel*numKernels)
      return ERROR_INCOMPATIBLE_DIMENSIONS ;


   int sharedMemSize = 4*(2*numPtsPerBlock+kernelWidth-1) ;
   if (sharedMemSize > 16*1024)
      throw "Specified parameters require kernel with shared memory greater than 16KB. Exiting" ; 
   dim3 gDim(numKernels, numBlocksPerKernel, 1) ; 
   KernPartialConvolve<<<gDim,numThreadsPerBlock,sharedMemSize>>>(signal1->data_device, 
                                                                  signal2->data_device, 
                                                                  scratchPad->data_device,
                                                                  signalLength, 
                                                                  kernelWidth, 
                                                                  numPtsPerBlock) ; 

   dim3 gDimSum(numKernels, kernelWidth,1) ; 
   KernPartialConvolveSum<<<gDimSum, numThreadsPerBlock, sizeof(float)*numThreadsPerBlock>>>(scratchPad->data_device,
			target->data_device, kernelWidth, numBlocksPerKernel, numKernels) ; 

   hipDeviceSynchronize();

   if (checkCUDAError())
      return CUDA_ERROR;
   return 0 ; 
}

// Use this convolution only when kernelWidth is small compared to signalLength because it 
// does one convolution per thread. If both are long, consider coding (/using NVIDIA's fft sample)
// with fft coefficient products.
extern int Convolve(cudamat *signal, cudamat *kernels, cudamat *target)
{
   if (!signal->on_device || !target->on_device || !kernels->on_device)
      return ERROR_NOT_ON_DEVICE;

   if (signal->size[0] != 1 && signal->size[1] != 1)
      return ERROR_INCOMPATIBLE_DIMENSIONS ;

   int signalLength = signal->size[0] * signal->size[1] ;
   int kernelWidth = kernels->size[0] ; 
   int numKernels = kernels->size[1] ; 

   if (target->size[0] != signalLength ||  target->size[1] != numKernels)
      return ERROR_INCOMPATIBLE_DIMENSIONS ;


   if (signal->is_trans)
      return ERROR_TRANSPOSED ;
   if (kernels->is_trans)
      return ERROR_TRANSPOSED ;
   if (target->is_trans)
      return ERROR_TRANSPOSED ;


   // Do calculation on device:  
   int block_size = 128 ;  
   int numPtsPerBlock = 128 ; 
   int numBlocksPerSignal = signalLength/numPtsPerBlock + (signalLength%numPtsPerBlock == 0 ? 0:1); 
   dim3 gridD(numKernels, numBlocksPerSignal,1) ; 
   int sharedMemSize = sizeof(float)*((numPtsPerBlock+kernelWidth-1) + kernelWidth) ; 

   KernConvolve <<< gridD, block_size, sharedMemSize >>>(signal->data_device, 
                                          kernels->data_device, 
                                          target->data_device, 
                                          signalLength, 
                                          kernelWidth, 
                                          numPtsPerBlock) ; 

   hipDeviceSynchronize();

   if (checkCUDAError())
      return CUDA_ERROR;

   return 0 ; 

}

extern int ReverseConvolve(cudamat *convolvedSignals, cudamat *kernels, cudamat * reverseConvolvedSignals)
{
   int kernelWidth = kernels->size[0] ; 
   int numKernels = kernels->size[1] ; 
   int signalLength = convolvedSignals->size[0] ;

   if (!convolvedSignals->on_device || !kernels->on_device || !reverseConvolvedSignals->on_device)
      return ERROR_NOT_ON_DEVICE;

   if (reverseConvolvedSignals->size[0] != signalLength || reverseConvolvedSignals->size[1] != numKernels)
      return ERROR_INCOMPATIBLE_DIMENSIONS ;

   if (convolvedSignals->size[1] !=  numKernels)
      return ERROR_INCOMPATIBLE_DIMENSIONS ;


   // Do calculation on device:  
   int numThreadsPerBlock = 32 ;  
   int numPtsPerThread = 1 ; 
   int numPtsPerBlock = numThreadsPerBlock*numPtsPerThread ; 
   int numBlocks = signalLength/numPtsPerBlock + (signalLength%numPtsPerBlock == 0 ? 0:1); 
   dim3 gridD(numKernels, numBlocks,1) ; 

   KernReverseConvolve <<< gridD, numThreadsPerBlock >>>(convolvedSignals->data_device, 
                                                     kernels->data_device, 
                                                     reverseConvolvedSignals->data_device, 
                                                     signalLength, kernelWidth, numKernels, 
                                                     numPtsPerBlock, numPtsPerThread) ; 

   hipDeviceSynchronize();

   if (checkCUDAError())
      return CUDA_ERROR;

   return 0 ; 
}

extern int Reconstruct(cudamat *convolvedSignals, 
                       cudamat *kernels, 
                       cudamat *reverseConvolvedSignals, 
                       cudamat *reconstruction)
{
   int kernelWidth = kernels->size[0] ; 
   int numKernels = kernels->size[1] ; 
   int signalLength = convolvedSignals->size[0] ;

   if (!convolvedSignals->on_device || !kernels->on_device || !reverseConvolvedSignals->on_device)
      return ERROR_NOT_ON_DEVICE;

   if (reverseConvolvedSignals->size[0]*reverseConvolvedSignals->size[1] < signalLength * numKernels)
      return ERROR_INCOMPATIBLE_DIMENSIONS ;

   if (convolvedSignals->size[1] !=  numKernels)
      return ERROR_INCOMPATIBLE_DIMENSIONS ;


   // Do calculation on device:  
   int numThreadsPerBlock = 256 ;  
   int numPtsPerThread = 1 ; 
   int numPtsPerBlock = numThreadsPerBlock*numPtsPerThread ; 
   int numBlocks = signalLength/numPtsPerBlock + (signalLength%numPtsPerBlock == 0 ? 0:1); 
   dim3 gridD(numKernels, numBlocks,1) ; 

   KernReverseConvolve <<< gridD, numThreadsPerBlock >>>(convolvedSignals->data_device, 
                                                     kernels->data_device, 
                                                     reverseConvolvedSignals->data_device, 
                                                     signalLength, kernelWidth, numKernels, 
                                                     numPtsPerBlock, numPtsPerThread) ; 

   KernAddSignals <<< numBlocks, numThreadsPerBlock >>>(reverseConvolvedSignals->data_device, 
                                                    reconstruction->data_device,
                                                    signalLength, 
                                                    numKernels, 
                                                    numPtsPerBlock, 
                                                    numPtsPerThread) ; 

   hipDeviceSynchronize();

   if (checkCUDAError())
      return CUDA_ERROR;

   return 0 ; 
}

}
